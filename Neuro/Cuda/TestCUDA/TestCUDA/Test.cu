#include<hip/hip_runtime.h>
#include<iostream>
#include<ctime>
#include<thread>

__global__ void kernel() {
	long long temp = 1000000000000;

	while (temp != 0) {
		temp--;
	}
}

void one() {
	long long temp = 1000000000000;

	while (temp != 1) {
		temp--;
	}
}

int main(void) {
	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Cuda failed!");
		return 1;
	}
	long start = clock();
	for (long i = 0; i < 1200000; i++) {
		kernel <<<1, 896 >>> ();
	}
	long end = clock();
	long temp = end - start;
	std::cout << temp << "   ms  896 Threads on GPU\n";
	start = clock();
	for (int i = 0; i < 8960; i++) {
		std::thread thr1(one);
		std::thread thr2(one);
		std::thread thr3(one);
		std::thread thr4(one);
		std::thread thr5(one);
		std::thread thr6(one);
		std::thread thr7(one);
		std::thread thr8(one);
		std::thread thr9(one);
		std::thread thr11(one);
		std::thread thr12(one);
		thr1.join();
		thr2.join();
		thr3.join();
		thr4.join();
		thr5.join();
		thr6.join();
		thr7.join();
		thr8.join();
		thr9.join();
		thr11.join();
		thr12.join();
	}

	end = clock();
	long temp1 = (end - start) * 100000;
	std::cout << temp1 << "    ms  12 Threads by CPU\n\n";

	std::cout << temp1 / temp << "  Times faster\n\n\n\n";
	std::cout << "Task: Do  " << 1000000000000 << "  subtractions  " << (1200000 * 896) << "  times\n\n\n\n\n";
	std::cin >> temp; // Что бы консоль не закрывалась если вы запускаете .exe
}
